#include "hip/hip_runtime.h"
/*************************************************************************
    > Author: Huang Di
    > Mail: hd232508@163.com 
    > Created Time: Fri 19 Mar 2021 07:42:04 PM CST
 ************************************************************************/
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// includes, project
//#include <cutil_inline.h>
//#include <helper_functions.h>

// includes, kernels
#include "matmul.cu"

////////////////////////////////////////////////////////////////////////////////
void runTest(int argc, char** argv);
void randomInit(float*, int);
void printDiff(float*, float*, int, int);

extern "C"
void computeGold(float*, const float*, const float*, unsigned int, unsigned int, unsigned int);

void
computeGold(float* C, const float* A, const float* B, unsigned int hA, unsigned int wA, unsigned int wB)
{
    for (unsigned int i = 0; i < hA; ++i)
        for (unsigned int j = 0; j < wB; ++j) {
            double sum = 0;
            for (unsigned int k = 0; k < wA; ++k) {
                double a = A[i * wA + k];
                double b = B[k * wB + j];
                sum += a * b;
            }
            C[i * wB + j] = (float)sum;
        }
}



////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main(int argc, char** argv)
{
    runTest(argc, argv);

    exit(EXIT_SUCCESS);
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
void
runTest(int argc, char** argv)
{

    /****************************************************/
    /*  Preparations                                    */
    /****************************************************/


    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = WA * HA;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);
    unsigned int size_B = WB * HB;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    float flop = 2 * (float)WC * (float)HC * (float)WA;
    printf("flops: %f\n", flop);

    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // allocate device memory for result
    unsigned int size_C = WC * HC;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

#if CHECK_RESULT == 1
    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // compute reference solution
    float* reference = (float*) malloc(mem_size_C);
    computeGold(reference, h_A, h_B, HA, WA, WB);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Naive CPU (Golden Reference)\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#endif

    dim3 threads,grid;

    /****************************************************/
    /*  Loop Unrolling                                  */
    /****************************************************/

    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    threads = dim3(BLOCK_SIZE, 4);
    grid = dim3(WC / (BLOCK_SIZE*4), HC / BLOCK_SIZE);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // naive implementation
    matrixMul_unroll<<< grid, threads >>>(d_C, d_A, d_B, WA, WB);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Loop unrolling GPU\n");
    printf("Processing time: %f (ms), GFLOPS: %f \n", msecTotal, flop / msecTotal/ 1e+6);
#if CHECK_RESULT == 1
    // check result
    printDiff(reference, h_C, WC, HC);
#endif

    /****************************************************/
    /*  Cleaning                                        */
    /****************************************************/

    // clean up memory
    free(h_A);
    free(h_B);
    free(h_C);
#if CHECK_RESULT == 1
    free(reference);
#endif
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

}

// Allocates a matrix with random float entries.
void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

void printDiff(float *data1, float *data2, int width, int height)
{
  int i,j,k;
  int error_count=0;
  for (j=0; j<height; j++) {
    for (i=0; i<width; i++) {
      k = j*width+i;
      if (fabs(data1[k] - data2[k]) > 0.1 ) {
         printf("diff(%d,%d) CPU=%4.4f, GPU=%4.4f \n", i,j, data1[k], data2[k]);
         error_count++;
      }
    }
  }
  printf("Total Errors = %d \n", error_count);
}


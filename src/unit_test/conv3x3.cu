#include "hip/hip_runtime.h"
#include <hipblas.h>
#include <iostream>
#include <stdio.h>
#include "conv_base.cu"
//time measure
#include <chrono>
#define CHECK_RESULT 1
#define MY 0
#define Batch 16
//#define BN 32
//#define BC 8
//#define BK 64
//#define BN 32*32
//#define BC 8*32
#define Bi 32    //input batch
#define Hi 128  //input h
#define Wi 128 //input w
#define BC 128 //input c
#define BK 64   //output c
#define PH 1    //pad h
#define PW 1    //pad w

void randomInit(float*, int);
void printDiff(float*, float*, int, int, int, int);

int main() {
    /****************************************************/
    /*  Preparations                                    */
    /****************************************************/
    
    int Ho=Hi-2+2*PH;
    int Wo=Wi-2+2*PW;
    int NH=(Ho+1)/2; //nH
    int NW=(Wo+1)/2; //nW
    int BN=NH*NW*Bi;  //N 

    hipEvent_t start;
    hipEvent_t stop;
    float msecTotal;

    // set seed for rand()
    srand(2006);

    // allocate host memory for matrices A and B
    unsigned int size_A = Bi * Hi * Wi * BC;
    unsigned int mem_size_A = sizeof(float) * size_A;
    float* h_A = (float*) malloc(mem_size_A);

    unsigned int size_B = 3 * 3 * BC * BK;
    unsigned int mem_size_B = sizeof(float) * size_B;
    float* h_B = (float*) malloc(mem_size_B);
    // initialize host memory
    randomInit(h_A, size_A);
    randomInit(h_B, size_B);
    
    // allocate device memory
    float* d_A;
    hipMalloc((void**) &d_A, mem_size_A);
    float* d_B;
    hipMalloc((void**) &d_B, mem_size_B);

    // allocate device memory for result
    unsigned int size_C = BK * Ho * Wo * Bi;
    unsigned int mem_size_C = sizeof(float) * size_C;
    float* d_C;
    hipMalloc((void**) &d_C, mem_size_C);

    // allocate host memory for the result
    float* h_C = (float*) malloc(mem_size_C);

//    int H_s[] = {0, 0, 0, 3, 3, 3, 6, 6, 6}; 
//    int H_e[] = {3, 3, 3, 6, 6, 6, 7, 7, 7}; 
//    int W_s[] = {0, 3, 6, 0, 3, 6, 0, 3, 6}; 
//    int W_e[] = {3, 6, 7, 3, 6, 7, 3, 6, 7}; 
//    int kernel_stride[] = {0, 16, 32, 40, 56, 72, 80, 88, 96};

    int H_s[] = {0}; 
    int H_e[] = {3}; 
    int W_s[] = {0}; 
    int W_e[] = {3}; 
    int kernel_stride[] = {0};

    const int num_split = sizeof(W_s) / sizeof(W_s[0]);
    const int N = num_split * BN * BK;

    int *H_start_gpu = nullptr;
    int *W_start_gpu = nullptr;
    int *H_end_gpu = nullptr;
    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(H_start_gpu, H_s, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_start_gpu, W_s, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_end_gpu, H_e, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_end_gpu, W_e, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    hipDeviceSynchronize();
#if MY == 1
    /****************************************************/
    /*  My kernel                                       */
    /****************************************************/

    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    // create and start timer
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
    // setup execution parameters
    // naive implementation
    int maxbytes = 67584; // 96 KB
    hipFuncSetAttribute(reinterpret_cast<const void*>(winograd2D<32), 8, 64>, hipFuncAttributeMaxDynamicSharedMemorySize, maxbytes);
    winograd2D<32, 8, 64><<<(BN/32)*(BK/64), 256, maxbytes>>>(d_A, d_C, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, NH, NW, Bi, Ho, Wo, BC, BK, PH, PW);
    // stop and destroy timer
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("My\n");
    printf("Processing time: %f (ms) \n", msecTotal);
    // copy result from device to host
    hipMemcpy(h_C, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);
    for(int i = 0; i < 10; i++) {
        cout << h_C[i] << " "; 
    }
    cout << endl;
    hipDeviceSynchronize();
#endif

#if CHECK_RESULT == 1
    /****************************************************/
    /*  Baseline                                        */
    /****************************************************/

    float* tmp_input_buffer;
    float* tmp_weight_buffer;
    float* tmp_product_buffer;
    float* tmp_out_buffer;
    int64_t* tmp_ptr_buffer;
    hipMalloc((void**) &tmp_input_buffer, Batch*Bi*NH*NW*BC*sizeof(float));
    hipMalloc((void**) &tmp_weight_buffer, Batch*BC*BK*sizeof(float));
    hipMalloc((void**) &tmp_product_buffer, Batch*Bi*NH*NW*BK*sizeof(float));
    hipMalloc((void**) &tmp_out_buffer, num_split*Bi*Ho*Wo*BK*sizeof(float));
    hipMalloc((void**) &tmp_ptr_buffer, 3*Batch*sizeof(int64_t));

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    // copy host memory to device
    hipMemcpy(d_A, h_A, mem_size_A,
                              hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, mem_size_B,
                              hipMemcpyHostToDevice);
    hipEventCreate(&start);
    hipEventRecord(start, NULL); 
////////////TODO: transform ///////////////
    convLauncherStrideOneLarge2D_base<float> (d_A, d_B, tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, tmp_ptr_buffer, Bi, Hi, Wi, BC, BK, 3, 3, PH, PW, d_C, num_split, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, tmp_out_buffer, handle);
///////////////////////////////////////////
    hipEventCreate(&stop);
    hipEventRecord(stop, NULL);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&msecTotal, start, stop);
    printf("Baseline\n");
    printf("Processing time: %f (ms) \n", msecTotal);
    float* ref = (float*) malloc(mem_size_C);
    hipMemcpy(ref, d_C, mem_size_C,
                              hipMemcpyDeviceToHost);

    for(int i = 0; i < 10; i++) {
        cout << ref[i] << " "; 
    }
    cout << endl;
    hipDeviceSynchronize();
#endif

    // check result
#if CHECK_RESULT == 1
//    printDiff(ref, h_C, BK, Ho, Wo, Bi);
    free(ref);
    hipFree(tmp_input_buffer);
    hipFree(tmp_weight_buffer);
    hipFree(tmp_product_buffer);
    hipFree(tmp_out_buffer);
    hipFree(tmp_ptr_buffer);
#endif
    free(h_A);
    free(h_B);
    free(h_C);
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    return 0;
}

void printDiff(float *data1, float *data2, int d0, int d1, int d2, int d3)
{
  int i,j,idx,l,x;
  int error_count=0;
    
  for (l=0; l<d0; l++) {
    for (j=0; j<d1; j++) {
      for (i=0; i<d2; i++) {
        for (x=0; x<d3; x++) {
          idx = l*d1*d2*d3+j*d2*d3+i*d3+x;
          if (fabs(data1[idx] - data2[idx]) > 0.00001 ) {
             printf("diff(%d,%d,%d,%d) CPU=%4.4f, GPU=%4.4f \n", l,j,i,x, data1[idx], data2[idx]);
             error_count++;
          }
        }
      }
    }
  }
  printf("Total Errors = %d \n", error_count);
}

void randomInit(float* data, int size)
{
    for (int i = 0; i < size; ++i)
        data[i] = rand() / (float)RAND_MAX;
}

#include "hip/hip_runtime.h"
/*************************************************************************
    > Author: Huang Di
    > Mail: hd232508@163.com 
    > Created Time: Thu 26 Nov 2020 04:16:33 PM CST
 ************************************************************************/
//#if GOOFLE_CUDA
//#define EIGEN_USE_GPU
#include <hipblas.h>
#include "../calculation_kernels2d.cu"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// dim3 threadsPerBlock(K)
// dim3 numBlocks(1, 1, C)
// norm_weight = (27, C, K)
// wino_weight = (64, C, K)
template <typename T>
__global__ void wNorm2WinoTransform2D(const T *norm_weight, T* wino_weight, const int *kernel_stride, const int *H_start, const int *H_end, const int *W_start, const int *W_end, int H, int W, int C, int K)
{
//    kernel_stride += s;
//    H_start += s;
//    W_start += s;
//    H_end += s;
//    W_end += s;

    int by = blockIdx.y; // n
    int c = blockIdx.x; // c
    int k = threadIdx.x; // k

    int splitH = H_end[by] - H_start[by];
    int splitW = W_end[by] - W_start[by];
    
//    TODO: need to +1
    T ele[9];

    for(int h = 0; h < splitH; h++) {
        for(int w = 0; w < splitW; w++) {
            ele[h * splitW + w] = norm_weight[(((h + H_start[by]) * W + w + W_start[by]) * C + c) * K + k];
        }
    }

    T product_weight_patch[16];

    wNorm2WinoCalculation2D(ele, product_weight_patch, splitH, splitW);


    for(int i = 0; i < (splitH+1)*(splitW+1); i++) {
//        product_weight_patch[i] = ele[i];
        wino_weight[(i + kernel_stride[by]) * C * K + c * K + k] = product_weight_patch[i];
    }
}

// dim3 threadsPerBlock(C, nH, 1)
// dim3 numBlocks(nW, nD, nB)

// I = (Batch, D, H, W, C)
// O = (64, Batch, nD, nH, nW, C)
template <typename T>
__global__ void inputNorm2WinoTransform2D(const T *norm_input, T *wino_input, const int *kernel_stride, const int *H_start, const int *H_end, const int *W_start, const int *W_end, int nH, int nW, int B, int H, int W, int C, int pad_h, int pad_w) {
    int bz = blockIdx.z; //n
    int by = blockIdx.y; //b
    int bx = blockIdx.x; //h*w
    int tx = threadIdx.x; //K
//    if(by*bx+tx == 0)
//    printf("inputNorm2WinoTransform called!!!!!!!!!!!!!!!!!\n");

    int h = bx / nW; 
    int w = bx % nW;

//    clock_t time_[9];
//    time_[0] = clock(); 

    int splitxH = H_end[bz] - H_start[bz] + 1;
    int splitxW = W_end[bz] - W_start[bz] + 1;

//    time_[1] = clock(); 

    int f_b = by;
    int xBase = 2 * w - pad_w;
    int yBase = 2 * h - pad_h;

//    time_[2] = clock(); 

    T input_patch[16] = {T(0)};
//    T *input_patch = new T[splitxD*splitxH*splitxW];
//    time_[3] = clock(); 


    int f_x, f_y;
      for(int j = 0; j < splitxH; j++) {
        for(int k = 0; k < splitxW; k++) {
          f_y = yBase + j + H_start[bz];
          f_x = xBase + k + W_start[bz];
          if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) {
            input_patch[j * splitxW + k] = norm_input[((f_b * H + f_y) * W + f_x) * C + tx];
          } else {
            input_patch[j * splitxW + k] = T(0);
          }
        }
      }
//    time_[4] = clock(); 

//    T *trans_input_patch = new T[splitxD*splitxH*splitxW];
    T trans_input_patch[16] = {T(0)};

//    time_[5] = clock(); 
//////// TODO: transformation functions here /////////////
//
//  __device__ function();
    inputNorm2WinoCalculation2D(input_patch, trans_input_patch, splitxH - 1, splitxW - 1);
//
//////////////////////////////////////////////////////////
//    time_[6] = clock(); 

    int offset = ((f_b * nH + h) * nW + w) * C + tx;
    int stride = B * nH * nW * C;

//    time_[7] = clock(); 
    for(int i = 0; i < splitxH*splitxW; i++) {
      wino_input[(i + kernel_stride[bz]) * stride + offset] = T(trans_input_patch[i]);
    }
//    time_[8] = clock(); 
//    for(int i = 0; i < 9 - 1; i++) {
//        time[i] = (int)(time_[i+1] - time_[i]);
//        time[i] = (int)(20);
//    }
}

template <typename T>
__global__ void inputNorm2WinoTransform2D2(const T *norm_input, T *wino_input, const int *kernel_stride, const int *H_start, const int *H_end, const int *W_start, const int *W_end, int nH, int nW, int B, int H, int W, int C, int pad_h, int pad_w, int N) {
//    kernel_stride += s;
//    H_start += s;
//    W_start += s;
//    H_end += s;
//    W_end += s;
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
    int bz = tid / (C * nH * nW * B); //n
    int by = (tid % (C * nH * nW * B)) / (C * nH * nW); //b
    int bx = (tid % (C * nH * nW * B) % (C * nH * nW)) / C; //h*w
    int tx = tid % (C * nH * nW * B) % (C * nH * nW) % C; //K
//    if(by*bx+tx == 0)
//    printf("inputNorm2WinoTransform called!!!!!!!!!!!!!!!!!\n");

    int h = bx / nW; 
    int w = bx % nW;

//    clock_t time_[9];
//    time_[0] = clock(); 

    int h_end = H_end[bz];
    int h_start = H_start[bz];
    int w_end = W_end[bz];
    int w_start = W_start[bz];
//    time_[1] = clock(); 

    int splitxH = h_end - h_start + 1;
    int splitxW = w_end - w_start + 1;
//    int splitxH = H_end[bz] - H_start[bz] + 1;
//    int splitxW = W_end[bz] - W_start[bz] + 1;


    int f_b = by;
    int xBase = 2 * w - pad_w;
    int yBase = 2 * h - pad_h;

//    time_[2] = clock(); 

    T input_patch[16];
//    T *input_patch = new T[splitxD*splitxH*splitxW];
//    time_[3] = clock(); 


    int f_x, f_y;
      for(int j = 0; j < splitxH; j++) {
        for(int k = 0; k < splitxW; k++) {
//          f_y = yBase + j + H_start[bz];
//          f_x = xBase + k + W_start[bz];
          f_y = yBase + j + h_start;
          f_x = xBase + k + w_start;
          if((f_x > -1) && (f_x < W) && (f_y > -1) && (f_y < H)) {
            input_patch[j * splitxW + k] = norm_input[((f_b * H + f_y) * W + f_x) * C + tx];
          } else {
            input_patch[j * splitxW + k] = T(0);
          }
        }
      }
//    time_[4] = clock(); 

////    T *trans_input_patch = new T[splitxD*splitxH*splitxW];
    T trans_input_patch[16];

//    time_[5] = clock(); 
//////// TODO: transformation functions here /////////////
//
//  __device__ function();
    inputNorm2WinoCalculation2D(input_patch, trans_input_patch, splitxH - 1, splitxW - 1);
//
//////////////////////////////////////////////////////////
//    time_[6] = clock(); 

    int offset = ((f_b * nH + h) * nW + w) * C + tx;
    int stride = B * nH * nW * C;
//
//    time_[7] = clock(); 
    for(int i = 0; i < splitxH*splitxW; i++) {
      wino_input[(i + kernel_stride[bz]) * stride + offset] = T(trans_input_patch[i]);
    }
//    time_[8] = clock(); 
//    if (tid == 0) {
//    for(int i = 0; i < 9 - 1; i++) {
//        time[i] = (int)(time_[i+1] - time_[i]);
//    }
//    }
    }
}

// dim3 threadsPerBlock(C, nH, 1)
// dim3 numBlocks(nW, nD, nB)
//wino_output = (64, Batch, nD, nH, nW, K)
//tmp_output = (Batch, D, H, W, K)
template <typename T>
__global__ void outputWino2NormTransform2D(const T *wino_output, T *tmp_output, const int *kernel_stride,  const int *H_start, const int *H_end, const int *W_start, const int *W_end, int B, int output_H, int output_W, int K, int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
//    clock_t time_[9];
    if (tid < N) {
//    time_[0] = clock(); 
    int nH, nW;
    nH = (output_H + 1) / 2;
    nW = (output_W + 1) / 2;
//    tmp_output += s * B * output_H * output_W * K;
//    kernel_stride += s;
///    H_start += s;
//    W_start += s;
//    H_end += s;
//    W_end += s;
//    int bz = blockIdx.z; //n
//    int by = blockIdx.y; //b
//    int bx = blockIdx.x; //h*w
//    int tx = threadIdx.x; //K
    int bz = tid / (K * nH * nW * B); //n
    int by = (tid % (K * nH * nW * B)) / (K * nH * nW); //b
    int bx = (tid % (K * nH * nW * B) % (K * nH * nW)) / K; //h*w
    int tx = tid % (K * nH * nW * B) % (K * nH * nW) % K; //K

    int h = bx / nW; 
    int w = bx % nW;

//    time_[1] = clock(); 

    int splitxH = H_end[bz] - H_start[bz] + 1;
    int splitxW = W_end[bz] - W_start[bz] + 1;

//    time_[2] = clock();

    T product_patch[16] = {0};

//    time_[3] = clock();

    product_patch[0] = wino_output[((((0 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[1] = wino_output[((((1 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[2] = wino_output[((((2 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[3] = wino_output[((((3 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[4] = wino_output[((((4 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[6] = wino_output[((((6 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[7] = wino_output[((((7 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[8] = wino_output[((((8 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[9] = wino_output[((((9 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[10] = wino_output[((((10 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[11] = wino_output[((((11 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[12] = wino_output[((((12 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[13] = wino_output[((((13 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[14] = wino_output[((((14 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    product_patch[15] = wino_output[((((15 + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
//    for(int i = 0; i < splitxH*splitxW; i++) {
//      product_patch[i] = wino_output[((((i + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
//    }

//    time_[4] = clock(); 

    T output_patch[4] = {0};

//    time_[5] = clock(); 

//////// TODO: transformation functions here /////////////
//
//  __device__ function();
    outputWino2NormCalculation2D(product_patch, output_patch, splitxH - 1, splitxW - 1);
//
//////////////////////////////////////////////////////////

//    time_[6] = clock();

    tmp_output[(((bz * B + by) * output_H + (2 * h + 0)) * output_W + (2 * w + 0)) * K + tx] = output_patch[0];
    if(output_W % 2 == 0 || w != nW - 1)
      tmp_output[(((bz * B + by) * output_H + (2 * h + 0)) * output_W + (2 * w + 1)) * K + tx] = output_patch[1];
    if(output_H % 2 == 0 || h != nH - 1)
      tmp_output[(((bz * B + by) * output_H + (2 * h + 1)) * output_W + (2 * w + 0)) * K + tx] = output_patch[2];
    if((output_W % 2 == 0 || w != nW - 1) && (output_H % 2 == 0 || h != nH - 1))
      tmp_output[(((bz * B + by) * output_H + (2 * h + 1)) * output_W + (2 * w + 1)) * K + tx] = output_patch[3];
    }
//    time_[7] = clock();
//    if (tid == 0) {
//    for(int i = 0; i < 7; i++) {
//        time[i] = (int)(time_[i+1] - time_[i]);
//    }
//    for(int i = 7; i < 9; i++) {
//        time[i] = 0; 
//    }
//    }
}

// dim3 threadsPerBlock(C, nH, 1)
// dim3 numBlocks(nW, nD, nB)
//tmp_output = (64, Batch, nD, nH, nW, K)
//norm_output = (Batch, D, H, W, K)
template <typename T>
__global__ void outputAggregate2D(T *tmp_output, T *norm_output, int numSplit, int B, int output_H, int output_W, int K, int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
//    int by = blockIdx.y; //b
//    int bx = blockIdx.x; //h*w
//    int tx = threadIdx.x; //K
//    if(by*bx+tx == 0)
//    printf("outputAggregate called!!!!!!!!!!!!!!!!!\n");
    int by = tid / (K * output_H * output_W); //n
    int bx = (tid % (K * output_H * output_W)) / K; //h*w
    int tx = tid % (K * output_H * output_W) % K; //K

    int h = bx / output_W;
    int w = bx % output_W;

    T result = (T)0;

    for(int i = 0; i < numSplit; i++) {
      result += tmp_output[(((i * B + by) * output_H + h) * output_W + w) * K + tx];
    }

    norm_output[((by * output_H + h) * output_W + w) * K + tx] = result;
    }
}

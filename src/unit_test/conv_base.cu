#include "hip/hip_runtime.h"
/*************************************************************************
    > Author: Huang Di
    > Mail: hd232508@163.com 
    > Created Time: Sun 28 Mar 2021 05:43:25 PM CST
 ************************************************************************/
#include <hipblas.h>
#include "../base_conv_launchers.h"
#include "../utils.cu"
#include <iostream>
#include <stdio.h>
#include "transform2d.cu"
//time measure
#include <chrono>

//#ifdef CUBLAS_API_H_
//static const char *_cudaGetErrorEnum(hipblasStatus_t error)
//{
//    switch (error)
//    {
//        case HIPBLAS_STATUS_SUCCESS:
//            return "HIPBLAS_STATUS_SUCCESS";
//
//        case HIPBLAS_STATUS_NOT_INITIALIZED:
//            return "HIPBLAS_STATUS_NOT_INITIALIZED";
//
//        case HIPBLAS_STATUS_ALLOC_FAILED:
//            return "HIPBLAS_STATUS_ALLOC_FAILED";
//
//        case HIPBLAS_STATUS_INVALID_VALUE:
//            return "HIPBLAS_STATUS_INVALID_VALUE";
//
//        case HIPBLAS_STATUS_ARCH_MISMATCH:
//            return "HIPBLAS_STATUS_ARCH_MISMATCH";
//
//        case HIPBLAS_STATUS_MAPPING_ERROR:
//            return "HIPBLAS_STATUS_MAPPING_ERROR";
//
//        case HIPBLAS_STATUS_EXECUTION_FAILED:
//            return "HIPBLAS_STATUS_EXECUTION_FAILED";
//
//        case HIPBLAS_STATUS_INTERNAL_ERROR:
//            return "HIPBLAS_STATUS_INTERNAL_ERROR";
//    }
//
//    return "<unknown>";
//}
//#endif
//#define checkCudaErrors( a ) do { \
//if (hipSuccess != (a)) { \
//f//printf(stderr, "Cuda runtime error in line %d of file %s \
//: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
//exit(EXIT_FAILURE); \
//} \
//} while(0);
//#define checkCudaErrors2( a ) do { \
//if (HIPBLAS_STATUS_SUCCESS != (a)) { \
//f//printf(stderr, "Cuda runtime error in line %d of file %s \
//: %s \n", __LINE__, __FILE__, _cudaGetErrorEnum(a) ); \
//exit(EXIT_FAILURE); \
//} \
//} while(0);
using namespace std;

template <typename T>
__global__ void outputWino2NormTransform2D_permute(const T *wino_output, T *tmp_output, const int *kernel_stride,  const int *H_start, const int *H_end, const int *W_start, const int *W_end, int B, int output_H, int output_W, int K, int N) {
    unsigned int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N) {
    int nH, nW;
    nH = (output_H + 1) / 2;
    nW = (output_W + 1) / 2;
    int bz = tid / (K * nH * nW * B); //n
    int by = (tid % (K * nH * nW * B)) / (K * nH * nW); //b
    int bx = (tid % (K * nH * nW * B) % (K * nH * nW)) / K; //h*w
    int tx = tid % (K * nH * nW * B) % (K * nH * nW) % K; //K

    int h = bx / nW; 
    int w = bx % nW;

    int splitxH = H_end[bz] - H_start[bz] + 1;
    int splitxW = W_end[bz] - W_start[bz] + 1;

    T product_patch[16] = {0};

    for(int i = 0; i < splitxH*splitxW; i++) {
      product_patch[i] = wino_output[((((i + kernel_stride[bz]) * B + by) * nH + h) * nW + w) * K + tx];
    }

    T output_patch[4] = {0};

    outputWino2NormCalculation2D(product_patch, output_patch, splitxH - 1, splitxW - 1);

    tmp_output[(((bz * B + tx) * output_H + (2 * h + 0)) * output_W + (2 * w + 0)) * B + by] = output_patch[0];
    if(output_W % 2 == 0 || w != nW - 1)
      tmp_output[(((bz * B + tx) * output_H + (2 * h + 0)) * output_W + (2 * w + 1)) * B + by] = output_patch[1];
    if(output_H % 2 == 0 || h != nH - 1)
      tmp_output[(((bz * B + tx) * output_H + (2 * h + 1)) * output_W + (2 * w + 0)) * B + by] = output_patch[2];
    if((output_W % 2 == 0 || w != nW - 1) && (output_H % 2 == 0 || h != nH - 1))
      tmp_output[(((bz * B + tx) * output_H + (2 * h + 1)) * output_W + (2 * w + 1)) * B + by] = output_patch[3];
    }
}

template <>
void convLauncherStrideOneLarge2D_base<float>(const float *input, const float *weight,
                              float *tmp_input_buffer, float *tmp_weight_buffer,
                              float *tmp_product_buffer, const int64_t *tmp_ptr_buffer,
                              int B, int H, int W, int C, int K,
                              int kernel_H, int kernel_W, int pad_h, int pad_w,
                              float *output,
                              int num_split, int *H_start_gpu, int *H_end_gpu, int *W_start_gpu, int *W_end_gpu, float *tmp_out_buffer, hipblasHandle_t handle)
{
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

//////////////////// a large LUT ///////////////////////
    int num_split2;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    splitControl2D(kernel_H, kernel_W, &num_split2, &H_start, &H_end, &W_start, &W_end); 

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
    }
    int kernel_size = kernel_stride[num_split-1] + (H_end[num_split-1] - H_start[num_split-1] + 1) * (W_end[num_split-1] - W_start[num_split-1] + 1);

    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    int N;
//    hipblasHandle_t handle = at::cuda::getCurrentCUDABlasHandle();

    N = C * nH * nW * B * num_split;
    cout << "N: " << N << endl;
    cout << "kernel_size: " << kernel_size << endl;
    inputNorm2WinoTransform2D2 <float> <<<(N - 1 + 512) / 512, 512>>> (input, tmp_input_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nH, nW, B, H, W, C, pad_h, pad_w, N);

    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
    wNorm2WinoTransform2D <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_H, kernel_W, C, K);

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size*2);

    dim3 bDim3(kernel_size, 1, 1);
    dim3 gDim3(1, 1, 1);
    forwardAssign2D <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nH, nW, K);

    float one = 1;
    float zero = 0;
  
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size);

    N = num_split*B*nH*nW*K;
    outputWino2NormTransform2D_permute <float> <<<(N - 1 + 512) / 512, 512>>> (tmp_product_buffer, tmp_out_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_H, output_W, K, N);

    N = B*output_H*output_W*K;
    outputAggregate2D<float> <<<(N - 1 + 512) / 512, 512>>> (tmp_out_buffer, output, num_split, B, output_H, output_W, K, N);

    hipFree(kernel_stride_gpu);
    delete[] H_start;
    delete[] W_start;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;
}

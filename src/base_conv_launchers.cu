#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <hipblas.h>
#include "src/base_conv_launchers.h"
#include "src/utils.cu"
#include <iostream>
//#include "utils.cu.cc"
#include <stdio.h>
#include "transform.cu"
#include "transform2d.cu"
//time measure
#include <chrono>
//#ifdef CUBLAS_API_H_
//static const char *_cudaGetErrorEnum(hipblasStatus_t error)
//{
//    switch (error)
//    {
//        case HIPBLAS_STATUS_SUCCESS:
//            return "HIPBLAS_STATUS_SUCCESS";
//
//        case HIPBLAS_STATUS_NOT_INITIALIZED:
//            return "HIPBLAS_STATUS_NOT_INITIALIZED";
//
//        case HIPBLAS_STATUS_ALLOC_FAILED:
//            return "HIPBLAS_STATUS_ALLOC_FAILED";
//
//        case HIPBLAS_STATUS_INVALID_VALUE:
//            return "HIPBLAS_STATUS_INVALID_VALUE";
//
//        case HIPBLAS_STATUS_ARCH_MISMATCH:
//            return "HIPBLAS_STATUS_ARCH_MISMATCH";
//
//        case HIPBLAS_STATUS_MAPPING_ERROR:
//            return "HIPBLAS_STATUS_MAPPING_ERROR";
//
//        case HIPBLAS_STATUS_EXECUTION_FAILED:
//            return "HIPBLAS_STATUS_EXECUTION_FAILED";
//
//        case HIPBLAS_STATUS_INTERNAL_ERROR:
//            return "HIPBLAS_STATUS_INTERNAL_ERROR";
//    }
//
//    return "<unknown>";
//}
//#endif
//#define checkCudaErrors( a ) do { \
//if (hipSuccess != (a)) { \
//f//printf(stderr, "Cuda runtime error in line %d of file %s \
//: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
//exit(EXIT_FAILURE); \
//} \
//} while(0);
//#define checkCudaErrors2( a ) do { \
//if (HIPBLAS_STATUS_SUCCESS != (a)) { \
//f//printf(stderr, "Cuda runtime error in line %d of file %s \
//: %s \n", __LINE__, __FILE__, _cudaGetErrorEnum(a) ); \
//exit(EXIT_FAILURE); \
//} \
//} while(0);
//#include "333.cu.cc"
//#include "333_2.cu.cc"
using namespace std;

template <>
void convLauncherStrideOneLarge<float>(const float *input, const float *weight,
                              float *tmp_input_buffer, float *tmp_weight_buffer,
                              float *tmp_product_buffer, const int64_t *tmp_ptr_buffer,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w,
                              float *output)
{
    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nD = (output_D + 1) / 2;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

//////////////////// a large LUT ///////////////////////
    int num_split;
    int *D_start = nullptr;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *D_end = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    splitControl(kernel_D, kernel_H, kernel_W, &num_split, &D_start, &D_end, &H_start, &H_end, &W_start, &W_end); 

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (D_end[i-1] - D_start[i-1] + 1) * (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
//        cout << kernel_stride[i] << endl;
    }

    int kernel_size = kernel_stride[num_split-1] + (D_end[num_split-1] - D_start[num_split-1] + 1) * (H_end[num_split-1] - H_start[num_split-1] + 1) * (W_end[num_split-1] - W_start[num_split-1] + 1);
//    cout << "kernel_size: " << kernel_size << endl;

    int *D_start_gpu = nullptr;
    int *H_start_gpu = nullptr;
    int *W_start_gpu = nullptr;
    int *D_end_gpu = nullptr;
    int *H_end_gpu = nullptr;
    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&D_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&D_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(D_start_gpu, D_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_start_gpu, H_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_start_gpu, W_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_end_gpu, D_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_end_gpu, H_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_end_gpu, W_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    dim3 bDim1(C, 1, 1);
    dim3 gDim1(nD*nH*nW, B, num_split);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
//    cout << "---------------------" << endl;
//    cout << B << D << H << W << C << K << num_split << endl;
//    cout << output_D << " " << output_H << " " << output_W << endl;
//    cout << "---------------------" << endl;
    inputNorm2WinoTransform <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nD, nH, nW, B, D, H, W, C, pad_d, pad_h, pad_w);
    wNorm2WinoTransform <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_D, kernel_H, kernel_W, C, K);

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size * 2);

    dim3 bDim3(kernel_size, 1, 1);
    dim3 gDim3(1, 1, 1);
    forwardAssign <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nD, nH, nW, K);

    float one = 1;
    float zero = 0;
  
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nD * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size);

//    cout << K << endl;
//    cout << C << endl;
//    cout << B * nD * nH * nW << endl;
//
    dim3 bDim4(K, 1, 1);
    dim3 gDim4(nD*nH*nW, B, num_split);
    float *tmp_output = nullptr;
    hipMalloc((void**)&tmp_output, num_split*B*output_D*output_H*output_W*K*sizeof(float));
    outputWino2NormTransform <float> <<<gDim4, bDim4>>> (tmp_product_buffer, tmp_output, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_D, output_H, output_W, K);

    dim3 bDim5(K, 1, 1);
    dim3 gDim5(output_D*output_H*output_W, B, 1);
    outputAggregate<float> <<<gDim5, bDim5>>> (tmp_output, output, num_split, B, output_D, output_H, output_W, K);

    hipblasDestroy(handle);
    hipFree(D_start_gpu);
    hipFree(H_start_gpu);
    hipFree(W_start_gpu);
    hipFree(D_end_gpu);
    hipFree(H_end_gpu);
    hipFree(W_end_gpu);
    hipFree(kernel_stride_gpu);
    hipFree(tmp_output);
    delete[] D_start;
    delete[] H_start;
    delete[] W_start;
    delete[] D_end;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;
}

template <>
void convLauncherStrideOneLarge2D<float>(const float *input, const float *weight,
                              float *tmp_input_buffer, float *tmp_weight_buffer,
                              float *tmp_product_buffer, const int64_t *tmp_ptr_buffer,
                              int B, int H, int W, int C, int K,
                              int kernel_H, int kernel_W, int pad_h, int pad_w,
                              float *output,
                              int num_split, int *H_start_gpu, int *H_end_gpu, int *W_start_gpu, int *W_end_gpu, float *tmp_out_buffer)
{
//  using std::chrono::high_resolution_clock;
//  using std::chrono:://duration_cast;
//  using std::chrono:://duration;
//  using std::chrono::milliseconds;
  //auto tt1 = high_resolution_clock::now();
//    //hipEvent_t t1, t2, t3, t4, t5, t6, t7, t8, t9, t10, t11, t12, t13, t14, t15, t16;
//    float elapsedTime;

    //hipEventCreate(&t1);
    //hipEventRecord(t1, 0);
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

//////////////////// a large LUT ///////////////////////
    int num_split2;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    splitControl2D(kernel_H, kernel_W, &num_split2, &H_start, &H_end, &W_start, &W_end); 

    //hipEventCreate(&t2);
    //hipEventRecord(t2,0);
    //hipEventSynchronize(t2);
  //auto tt2 = high_resolution_clock::now();
    //duration<double, std::milli> //ms_double = tt2 - tt1;
    //std::cout << "splitcontrol cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t1, t2);
    //printf("splitcontrol time : %f ms\n" ,elapsedTime);

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
//        cout << kernel_stride[i] << endl;
    }

    int kernel_size = kernel_stride[num_split-1] + (H_end[num_split-1] - H_start[num_split-1] + 1) * (W_end[num_split-1] - W_start[num_split-1] + 1);
//    cout << "kernel_size: " << kernel_size << endl;

    //hipEventCreate(&t3);
    //hipEventRecord(t3,0);
    //hipEventSynchronize(t3);
  //auto tt3 = high_resolution_clock::now();
    //ms_double = tt3 - tt2;
    //std::cout << "kernel_stride cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t2, t3);
    //printf("kernel_stride time: %f ms\n" ,elapsedTime);

//    int *H_start_gpu = nullptr;
//    int *W_start_gpu = nullptr;
//    int *H_end_gpu = nullptr;
//    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
//    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
//    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
//    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
//    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
//    hipMemcpy(H_start_gpu, H_start, num_split*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(W_start_gpu, W_start, num_split*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(H_end_gpu, H_end, num_split*sizeof(int), hipMemcpyHostToDevice);
//    hipMemcpy(W_end_gpu, W_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    //hipEventCreate(&t4);
    //hipEventRecord(t4,0);
    //hipEventSynchronize(t4);
  //auto tt4 = high_resolution_clock::now();
    //ms_double = tt4 - tt3;
    //std::cout << "malloc cpy cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t3, t4);
    //printf("malloc cpy time: %f ms\n" ,elapsedTime);

    dim3 bDim1(C, 1, 1);
    dim3 gDim1(nH*nW, B, num_split);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
    inputNorm2WinoTransform2D <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nH, nW, B, H, W, C, pad_h, pad_w);

    //hipEventCreate(&t5);
    //hipEventRecord(t5,0);
    //hipEventSynchronize(t5);
  //auto tt5 = high_resolution_clock::now();
    //ms_double = tt5 - tt4;
    //std::cout << "input trans cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t4, t5);
    //printf("input trans time : %f ms\n" ,elapsedTime);

    wNorm2WinoTransform2D <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_H, kernel_W, C, K);

    //hipEventCreate(&t6);
    //hipEventRecord(t6,0);
    //hipEventSynchronize(t6);
  //auto tt6 = high_resolution_clock::now();
    //ms_double = tt6 - tt5;
    //std::cout << "w trans cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t5, t6);
    //printf("w trans time : %f ms\n" ,elapsedTime);

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size * 2);

    dim3 bDim3(kernel_size, 1, 1);
    dim3 gDim3(1, 1, 1);
    forwardAssign2D <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nH, nW, K);

    float one = 1;
    float zero = 0;
  
    //hipEventCreate(&t7);
    //hipEventRecord(t7,0);
    //hipEventSynchronize(t7);
  //auto tt7 = high_resolution_clock::now();
    //ms_double = tt7 - tt6;
    //std::cout << "forwardassign cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t6, t7);
    //printf("forwardassign time : %f ms\n" ,elapsedTime);

//    hipblasHandle_t handle;
//    handle = ;

    //hipEventCreate(&t14);
    //hipEventRecord(t14,0);
    //hipEventSynchronize(t14);
  //auto tt14 = high_resolution_clock::now();
    //ms_double = tt14 - tt7;
    //std::cout << "init handle cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t7, t14);
    //printf("init handle time : %f ms\n" ,elapsedTime);

//    hipblasCreate(&handle);

    //hipEventCreate(&t15);
    //hipEventRecord(t15,0);
    //hipEventSynchronize(t15);
  //auto tt15 = high_resolution_clock::now();
    //ms_double = tt15 - tt14;
    //std::cout << "create handle cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t14, t15);
    //printf("create handle time : %f ms\n" ,elapsedTime);
    hipblasSgemmBatched(at::cuda::getCurrentCUDABlasHandle(), HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size);

    //hipEventCreate(&t8);
    //hipEventRecord(t8,0);
    //hipEventSynchronize(t8);
  //auto tt8 = high_resolution_clock::now();
    //ms_double = tt8 - tt15;
    //std::cout << "gemm cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t15, t8);
    //printf("gemm time : %f ms\n" ,elapsedTime);

    dim3 bDim4(K, 1, 1);
    dim3 gDim4(nH*nW, B, num_split);
//    float *tmp_output = nullptr;
//    hipMalloc((void**)&tmp_output, num_split*B*output_H*output_W*K*sizeof(float));

    //hipEventCreate(&t16);
    //hipEventRecord(t16, 0);
    //hipEventSynchronize(t16);
  //auto tt16 = high_resolution_clock::now();
    //ms_double = tt16 - tt8;
    //std::cout << "out trans cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t8, t16);
    //printf("out trans time : %f ms\n" ,elapsedTime);

    outputWino2NormTransform2D <float> <<<gDim4, bDim4>>> (tmp_product_buffer, tmp_out_buffer, kernel_stride_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_H, output_W, K);

    //hipEventCreate(&t9);
    //hipEventRecord(t9,0);
    //hipEventSynchronize(t9);
  //auto tt9 = high_resolution_clock::now();
    //ms_double = tt9 - tt16;
    //std::cout << "out trans cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t16, t9);
    //printf("out trans time : %f ms\n" ,elapsedTime);

    dim3 bDim5(K, 1, 1);
    dim3 gDim5(output_H*output_W, B, 1);
    outputAggregate2D<float> <<<gDim5, bDim5>>> (tmp_out_buffer, output, num_split, B, output_H, output_W, K);
    //hipEventCreate(&t10);
    //hipEventRecord(t10,0);
    //hipEventSynchronize(t10);
  //auto tt10 = high_resolution_clock::now();
    //ms_double = tt10 - tt9;
    //std::cout << "aggregate cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t9, t10);
    //printf("aggregate time : %f ms\n" ,elapsedTime);


//    hipblasDestroy(handle);
    //hipEventCreate(&t11);
    //hipEventRecord(t11,0);
    //hipEventSynchronize(t11);
  //auto tt11 = high_resolution_clock::now();
    //ms_double = tt11 - tt10;
    //std::cout << "hipblasDestroy cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t10, t11);
    //printf("hipblasDestroy time: %f ms\n" ,elapsedTime);
//    hipFree(H_start_gpu);
//    hipFree(W_start_gpu);
//    hipFree(H_end_gpu);
//    hipFree(W_end_gpu);
    hipFree(kernel_stride_gpu);
//    hipFree(tmp_output);

    //hipEventCreate(&t12);
    //hipEventRecord(t12,0);
    //hipEventSynchronize(t12);
  //auto tt12 = high_resolution_clock::now();
    //ms_double = tt12 - tt11;
    //std::cout << "free cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t11, t12);
    //printf("free time: %f ms\n" ,elapsedTime);

    delete[] H_start;
    delete[] W_start;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;
    //hipEventCreate(&t13);
    //hipEventRecord(t13,0);
    //hipEventSynchronize(t13);
  //auto tt13 = high_resolution_clock::now();
    //ms_double = tt13 - tt12;
    //std::cout << "delete cpu: "<< //ms_double.count() << " ms" << endl;

    //hipEventElapsedTime(&elapsedTime, t12, t13);
    //printf("delete time: %f ms\n" ,elapsedTime);

}

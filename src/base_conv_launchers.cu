#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include <hipblas.h>
#include "src/base_conv_launchers.h"
#include "src/transform_kernels.cu"
#include "src/utils.cu"
#include <iostream>
//#include "utils.cu.cc"
#include <stdio.h>
#include "transform_kernels_3d.cu"
#include "split_kernel.cu"
#ifdef CUBLAS_API_H_
static const char *_cudaGetErrorEnum(hipblasStatus_t error)
{
    switch (error)
    {
        case HIPBLAS_STATUS_SUCCESS:
            return "HIPBLAS_STATUS_SUCCESS";

        case HIPBLAS_STATUS_NOT_INITIALIZED:
            return "HIPBLAS_STATUS_NOT_INITIALIZED";

        case HIPBLAS_STATUS_ALLOC_FAILED:
            return "HIPBLAS_STATUS_ALLOC_FAILED";

        case HIPBLAS_STATUS_INVALID_VALUE:
            return "HIPBLAS_STATUS_INVALID_VALUE";

        case HIPBLAS_STATUS_ARCH_MISMATCH:
            return "HIPBLAS_STATUS_ARCH_MISMATCH";

        case HIPBLAS_STATUS_MAPPING_ERROR:
            return "HIPBLAS_STATUS_MAPPING_ERROR";

        case HIPBLAS_STATUS_EXECUTION_FAILED:
            return "HIPBLAS_STATUS_EXECUTION_FAILED";

        case HIPBLAS_STATUS_INTERNAL_ERROR:
            return "HIPBLAS_STATUS_INTERNAL_ERROR";
    }

    return "<unknown>";
}
#endif
#define checkCudaErrors( a ) do { \
if (hipSuccess != (a)) { \
fprintf(stderr, "Cuda runtime error in line %d of file %s \
: %s \n", __LINE__, __FILE__, hipGetErrorString(hipGetLastError()) ); \
exit(EXIT_FAILURE); \
} \
} while(0);
#define checkCudaErrors2( a ) do { \
if (HIPBLAS_STATUS_SUCCESS != (a)) { \
fprintf(stderr, "Cuda runtime error in line %d of file %s \
: %s \n", __LINE__, __FILE__, _cudaGetErrorEnum(a) ); \
exit(EXIT_FAILURE); \
} \
} while(0);
//#include "333.cu.cc"
//#include "333_2.cu.cc"
using namespace std;

template <>
void split<float>(const float *input, const float *weight, const float *tmp_product_buffer,
                              float *tmp_input_buffer, float *tmp_weight_buffer, float *output,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w)
{
    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nD = (output_D + 1) / 2;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

    int *D_start = nullptr;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *D_end = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    int num_split;
    
    splitControl(kernel_D, kernel_H, kernel_W, &num_split, &D_start, &D_end, &H_start, &H_end, &W_start, &W_end); 

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (D_end[i-1] - D_start[i-1] + 1) * (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
//        cout << kernel_stride[i] << endl;
    }

    int *D_start_gpu = nullptr;
    int *H_start_gpu = nullptr;
    int *W_start_gpu = nullptr;
    int *D_end_gpu = nullptr;
    int *H_end_gpu = nullptr;
    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&D_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&D_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(D_start_gpu, D_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_start_gpu, H_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_start_gpu, W_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_end_gpu, D_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_end_gpu, H_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_end_gpu, W_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    dim3 bDim1(C, 1, 1);
    dim3 gDim1(nD*nH*nW, B, num_split);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
//    cout << B << D << H << W << C << endl;
//    inputNorm2WinoTransform <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nD, nH, nW, B, D, H, W, C, pad_d, pad_h, pad_w);
//    wNorm2WinoTransform <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_D, kernel_H, kernel_W, C, K);

//    dim3 bDim3(65, nH, num_split);
//    dim3 gDim3(nW, nD, B);
    dim3 bDim3(K, 1, 1);
    dim3 gDim3(nD*nH*nW, B, num_split);
//    float *tmp_output = nullptr;
//    hipMalloc((void**)&tmp_output, num_split*B*output_D*output_H*output_W*K*sizeof(float));
    outputWino2NormTransform <float> <<<gDim3, bDim3>>> (tmp_product_buffer, output, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_D, output_H, output_W, K);

//    dim3 bDim5(K, 1, 1);
//    dim3 gDim5(output_D*output_H*output_W, B, 1);
//    outputAggregate<float> <<<gDim4, bDim4>>> (tmp_output, output, num_split, B, output_D, output_H, output_W, K);
    hipFree(D_start_gpu);
    hipFree(H_start_gpu);
    hipFree(W_start_gpu);
    hipFree(D_end_gpu);
    hipFree(H_end_gpu);
    hipFree(W_end_gpu);
    hipFree(kernel_stride_gpu);
//    hipFree(tmp_output);
    delete[] D_start;
    delete[] H_start;
    delete[] W_start;
    delete[] D_end;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;
}

template <>
void convLauncherStrideOneLarge2<float>(const float *input, const float *weight,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w,
                              float *output)
{
    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nD = (output_D + 1) / 2;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

    int kernel_size1 = int((kernel_D + 1 + (kernel_D - 1) / 3) * (kernel_H + 1 + (kernel_H - 1) / 3) * (kernel_W + 1 + (kernel_W - 1) / 3));
    float *tmp_input_buffer = nullptr;
    float *tmp_weight_buffer = nullptr;
    float *tmp_product_buffer = nullptr;
    int64_t *tmp_ptr_buffer = nullptr;
    hipMalloc((void**)&tmp_input_buffer, kernel_size1*B*nD*nH*nW*C*sizeof(float));
    hipMalloc((void**)&tmp_weight_buffer, kernel_size1*C*K*sizeof(float));
    hipMalloc((void**)&tmp_product_buffer, kernel_size1*nD*nH*nW*B*K*sizeof(float));
    hipMalloc((void**)&tmp_ptr_buffer, 3*kernel_size1*sizeof(int64_t));

//////////////////// a large LUT ///////////////////////
    int num_split;
    int *D_start = nullptr;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *D_end = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    splitControl(kernel_D, kernel_H, kernel_W, &num_split, &D_start, &D_end, &H_start, &H_end, &W_start, &W_end); 

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (D_end[i-1] - D_start[i-1] + 1) * (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
//        cout << kernel_stride[i] << endl;
    }

    int kernel_size = kernel_stride[num_split-1] + (D_end[num_split-1] - D_start[num_split-1] + 1) * (H_end[num_split-1] - H_start[num_split-1] + 1) * (W_end[num_split-1] - W_start[num_split-1] + 1);
//    cout << "kernel_size: " << kernel_size << endl;

    int *D_start_gpu = nullptr;
    int *H_start_gpu = nullptr;
    int *W_start_gpu = nullptr;
    int *D_end_gpu = nullptr;
    int *H_end_gpu = nullptr;
    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&D_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&D_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(D_start_gpu, D_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_start_gpu, H_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_start_gpu, W_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_end_gpu, D_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_end_gpu, H_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_end_gpu, W_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    dim3 bDim1(C, 1, 1);
    dim3 gDim1(nD*nH*nW, B, num_split);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
//    cout << "---------------------" << endl;
//    cout << B << D << H << W << C << K << num_split << endl;
//    cout << output_D << " " << output_H << " " << output_W << endl;
//    cout << "---------------------" << endl;
    inputNorm2WinoTransform <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nD, nH, nW, B, D, H, W, C, pad_d, pad_h, pad_w);
    wNorm2WinoTransform <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_D, kernel_H, kernel_W, C, K);

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size * 2);

    dim3 bDim3(kernel_size, 1, 1);
    dim3 gDim3(1, 1, 1);
    forwardAssign <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nD, nH, nW, K);

    float one = 1;
    float zero = 0;
  
    hipblasHandle_t handle;
    checkCudaErrors2(hipblasCreate(&handle));
    checkCudaErrors2(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nD * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size));

//    cout << K << endl;
//    cout << C << endl;
//    cout << B * nD * nH * nW << endl;
//
    dim3 bDim4(K, 1, 1);
    dim3 gDim4(nD*nH*nW, B, num_split);
    float *tmp_output = nullptr;
    checkCudaErrors(hipMalloc((void**)&tmp_output, num_split*B*output_D*output_H*output_W*K*sizeof(float)));
    outputWino2NormTransform <float> <<<gDim4, bDim4>>> (tmp_product_buffer, tmp_output, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_D, output_H, output_W, K);

    dim3 bDim5(K, 1, 1);
    dim3 gDim5(output_D*output_H*output_W, B, 1);
    outputAggregate<float> <<<gDim5, bDim5>>> (tmp_output, output, num_split, B, output_D, output_H, output_W, K);

    checkCudaErrors2(hipblasDestroy(handle));
    hipFree(D_start_gpu);
    hipFree(H_start_gpu);
    hipFree(W_start_gpu);
    hipFree(D_end_gpu);
    hipFree(H_end_gpu);
    hipFree(W_end_gpu);
    hipFree(kernel_stride_gpu);
    hipFree(tmp_output);
    delete[] D_start;
    delete[] H_start;
    delete[] W_start;
    delete[] D_end;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;

    hipFree(tmp_input_buffer);
    hipFree(tmp_weight_buffer);
    hipFree(tmp_product_buffer);
    hipFree(tmp_ptr_buffer);
}

template <>
void convLauncherStrideOneLarge<float>(const float *input, const float *weight,
                              float *tmp_input_buffer, float *tmp_weight_buffer,
                              float *tmp_product_buffer, const int64_t *tmp_ptr_buffer,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w,
                              float *output)
{
    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
    int nD = (output_D + 1) / 2;
    int nH = (output_H + 1) / 2;
    int nW = (output_W + 1) / 2;

//////////////////// a large LUT ///////////////////////
    int num_split;
    int *D_start = nullptr;
    int *H_start = nullptr;
    int *W_start = nullptr;
    int *D_end = nullptr;
    int *H_end = nullptr;
    int *W_end = nullptr;
    splitControl(kernel_D, kernel_H, kernel_W, &num_split, &D_start, &D_end, &H_start, &H_end, &W_start, &W_end); 

    int *kernel_stride = new int[num_split]();
    for(int i = 1; i < num_split; i++) {
        kernel_stride[i] = (D_end[i-1] - D_start[i-1] + 1) * (H_end[i-1] - H_start[i-1] + 1) * (W_end[i-1] - W_start[i-1] + 1) + kernel_stride[i-1];
//        cout << kernel_stride[i] << endl;
    }

    int kernel_size = kernel_stride[num_split-1] + (D_end[num_split-1] - D_start[num_split-1] + 1) * (H_end[num_split-1] - H_start[num_split-1] + 1) * (W_end[num_split-1] - W_start[num_split-1] + 1);
//    cout << "kernel_size: " << kernel_size << endl;

    int *D_start_gpu = nullptr;
    int *H_start_gpu = nullptr;
    int *W_start_gpu = nullptr;
    int *D_end_gpu = nullptr;
    int *H_end_gpu = nullptr;
    int *W_end_gpu = nullptr;
    int *kernel_stride_gpu = nullptr;
    hipMalloc((void**)&D_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&D_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&H_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_start_gpu, num_split*sizeof(int));
    hipMalloc((void**)&W_end_gpu, num_split*sizeof(int));
    hipMalloc((void**)&kernel_stride_gpu, num_split*sizeof(int));
    hipMemcpy(D_start_gpu, D_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_start_gpu, H_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_start_gpu, W_start, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(D_end_gpu, D_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_end_gpu, H_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(W_end_gpu, W_end, num_split*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(kernel_stride_gpu, kernel_stride, num_split*sizeof(int), hipMemcpyHostToDevice);

    dim3 bDim1(C, 1, 1);
    dim3 gDim1(nD*nH*nW, B, num_split);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, num_split, 1);
//    cout << "---------------------" << endl;
//    cout << B << D << H << W << C << K << num_split << endl;
//    cout << output_D << " " << output_H << " " << output_W << endl;
//    cout << "---------------------" << endl;
    inputNorm2WinoTransform <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, nD, nH, nW, B, D, H, W, C, pad_d, pad_h, pad_w);
    wNorm2WinoTransform <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, kernel_D, kernel_H, kernel_W, C, K);

    const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
    const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
    float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size * 2);

    dim3 bDim3(kernel_size, 1, 1);
    dim3 gDim3(1, 1, 1);
    forwardAssign <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nD, nH, nW, K);

    float one = 1;
    float zero = 0;
  
    hipblasHandle_t handle;
    checkCudaErrors2(hipblasCreate(&handle));
    checkCudaErrors2(hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nD * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size));

//    cout << K << endl;
//    cout << C << endl;
//    cout << B * nD * nH * nW << endl;
//
    dim3 bDim4(K, 1, 1);
    dim3 gDim4(nD*nH*nW, B, num_split);
    float *tmp_output = nullptr;
    checkCudaErrors(hipMalloc((void**)&tmp_output, num_split*B*output_D*output_H*output_W*K*sizeof(float)));
    outputWino2NormTransform <float> <<<gDim4, bDim4>>> (tmp_product_buffer, tmp_output, kernel_stride_gpu, D_start_gpu, D_end_gpu, H_start_gpu, H_end_gpu, W_start_gpu, W_end_gpu, B, output_D, output_H, output_W, K);

    dim3 bDim5(K, 1, 1);
    dim3 gDim5(output_D*output_H*output_W, B, 1);
    outputAggregate<float> <<<gDim5, bDim5>>> (tmp_output, output, num_split, B, output_D, output_H, output_W, K);

    checkCudaErrors2(hipblasDestroy(handle));
    hipFree(D_start_gpu);
    hipFree(H_start_gpu);
    hipFree(W_start_gpu);
    hipFree(D_end_gpu);
    hipFree(H_end_gpu);
    hipFree(W_end_gpu);
    hipFree(kernel_stride_gpu);
    hipFree(tmp_output);
    delete[] D_start;
    delete[] H_start;
    delete[] W_start;
    delete[] D_end;
    delete[] H_end;
    delete[] W_end;
    delete[] kernel_stride;
}

template <>
void convLauncherStrideOne3x3<float>(const float *input, const float *weight,
                              float *tmp_input_buffer, float *tmp_weight_buffer,
                              float *tmp_product_buffer, const int64_t *tmp_ptr_buffer,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w,
                              float *output)
{
    int kernel_size = (kernel_D + 1) * (kernel_H + 1) * (kernel_W + 1);
    int nD = (D + 1 + 2 * pad_d - (kernel_D + 1)) / 2 + 1;
    int nH = (H + 1 + 2 * pad_h - (kernel_H + 1)) / 2 + 1;
    int nW = (W + 1 + 2 * pad_w - (kernel_W + 1)) / 2 + 1;

//      cout << kernel_D << kernel_H << kernel_W << endl;
//      cout << B << D << H << W  << C << K << endl;
//      cout << pad_d << pad_h << pad_w  << endl;
//    cout << nD << nH << nW << kernel_size << endl;

    dim3 bDim1(C, nH, 1);
    dim3 gDim1(nW, nD, B);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, 1, 1);

    if(kernel_D == 1 and kernel_H == 1 and kernel_W == 1) {
        inputNorm2WinoTransform1x1x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x1x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform1x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform1x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform1x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform1x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform1x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform1x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform1x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform1x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 1) {
        inputNorm2WinoTransform2x1x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform2x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform2x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform2x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform2x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform2x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform2x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform2x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform2x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 1) {
        inputNorm2WinoTransform3x1x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform3x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform3x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform3x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform3x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform3x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform3x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform3x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform3x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    }
	const float** Input_ptrs_gpu_ = (const float **)(tmp_ptr_buffer);
	const float** Weight_ptrs_gpu_ = (const float **)(tmp_ptr_buffer + kernel_size);
	float** tmp_product_ptrs_gpu_ = (float **)(tmp_ptr_buffer + kernel_size * 2);

	dim3 bDim3(kernel_size, 1, 1);
	dim3 gDim3(1, 1, 1);
	forwardAssign <float> <<<gDim3, bDim3>>> (tmp_input_buffer, tmp_weight_buffer, tmp_product_buffer, Input_ptrs_gpu_, Weight_ptrs_gpu_, tmp_product_ptrs_gpu_, C, B, nD, nH, nW, K);

	float one = 1;
	float zero = 0;
    
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	hipblasSgemmBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        K, B * nD * nH * nW, C,
        &one,
        Weight_ptrs_gpu_, K,
        Input_ptrs_gpu_, C,
        &zero, tmp_product_ptrs_gpu_, K, kernel_size);

    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
	dim3 blockDim(K, nH, 1);
	dim3 gridDim(nW, nD, B);
    if(kernel_D == 1 and kernel_H == 1 and kernel_W == 1) {
	    outputWino2NormTransform1x1x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform1x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform1x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform1x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform1x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform1x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform1x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform1x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform1x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 1) {
	    outputWino2NormTransform2x1x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform2x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform2x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform2x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform2x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform2x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform2x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform2x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform2x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 1) {
	    outputWino2NormTransform3x1x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform3x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform3x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform3x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform3x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform3x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform3x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform3x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform3x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    }

	hipblasDestroy(handle);
}

template <>
void transform<float>(const float *input, const float *weight, const float *tmp_product_buffer,
                              float *tmp_input_buffer, float *tmp_weight_buffer, float *output,
                              int B, int D, int H, int W, int C, int K,
                              int kernel_D, int kernel_H, int kernel_W, int pad_d, int pad_h, int pad_w)
{
    int nD = (D + 1 + 2 * pad_d - (kernel_D + 1)) / 2 + 1;
    int nH = (H + 1 + 2 * pad_h - (kernel_H + 1)) / 2 + 1;
    int nW = (W + 1 + 2 * pad_w - (kernel_W + 1)) / 2 + 1;
    dim3 bDim1(C, nH, 1);
    dim3 gDim1(nW, nD, B);
    dim3 bDim2(K, 1, 1);
    dim3 gDim2(C, 1, 1);
//    cout << B << D << H << W << C << endl;
    if(kernel_D == 1 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform1x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform1x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform1x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform1x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform1x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform1x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform1x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform1x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform1x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 1) {
        inputNorm2WinoTransform2x1x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform2x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform2x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform2x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform2x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform2x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform2x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform2x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform2x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform2x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 1) {
        inputNorm2WinoTransform3x1x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 2) {
        inputNorm2WinoTransform3x1x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 3) {
        inputNorm2WinoTransform3x1x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x1x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 1) {
        inputNorm2WinoTransform3x2x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 2) {
        inputNorm2WinoTransform3x2x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 3) {
        inputNorm2WinoTransform3x2x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x2x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 1) {
        inputNorm2WinoTransform3x3x1 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x1 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 2) {
        inputNorm2WinoTransform3x3x2 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x2 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 3) {
        inputNorm2WinoTransform3x3x3 <float> <<<gDim1, bDim1>>> (input, tmp_input_buffer, B, D, H, W, C, pad_d, pad_h, pad_w);
        wNorm2WinoTransform3x3x3 <float> <<<gDim2, bDim2>>> (weight, tmp_weight_buffer, C, K);
    }

    int output_D = (D + 2 * pad_d - kernel_D) / 1 + 1;
    int output_H = (H + 2 * pad_h - kernel_H) / 1 + 1;
    int output_W = (W + 2 * pad_w - kernel_W) / 1 + 1;
	dim3 blockDim(K, nH, 1);
	dim3 gridDim(nW, nD, B);
    if(kernel_D == 1 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform1x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform1x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform1x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform1x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform1x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform1x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform1x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 1 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform1x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 1) {
	    outputWino2NormTransform2x1x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform2x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform2x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform2x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform2x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform2x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform2x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform2x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 2 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform2x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 1) {
	    outputWino2NormTransform3x1x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 2) {
	    outputWino2NormTransform3x1x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 1 and kernel_W == 3) {
	    outputWino2NormTransform3x1x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 1) {
	    outputWino2NormTransform3x2x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 2) {
	    outputWino2NormTransform3x2x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 2 and kernel_W == 3) {
	    outputWino2NormTransform3x2x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 1) {
	    outputWino2NormTransform3x3x1 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 2) {
	    outputWino2NormTransform3x3x2 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    } else if(kernel_D == 3 and kernel_H == 3 and kernel_W == 3) {
	    outputWino2NormTransform3x3x3 <float> <<<gridDim, blockDim>>> (tmp_product_buffer, output, B, output_D, output_H, output_W, K);
    }
}
